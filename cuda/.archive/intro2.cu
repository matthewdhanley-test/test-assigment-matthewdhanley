
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>

// This program has been adapted from NVIDIA's "An Even Easier Introduction
// to CUDA." All credit goes to Mark Harris

// https://devblogs.nvidia.com/even-easier-introduction-cuda/

// function to add the elements of two arrays
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

__global__
void d_add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int add_cpu(void){
  int N = 1<<20; // 1M elements
  printf("Number of elements: %d\n",N);

  float *x = new float[N];
  float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  clock_t begin = clock();
  add(N, x, y);
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("CPU Time: %f seconds\n", time_spent);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete [] x;
  delete [] y;

  return 0;
}

int add_gpu(void){

  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  clock_t begin = clock();

  d_add<<<numBlocks, blockSize>>>(N, x, y);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("GPU Time: %f seconds\n", time_spent);


  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}


int main(void)
{
  add_cpu();
  add_gpu();
}
